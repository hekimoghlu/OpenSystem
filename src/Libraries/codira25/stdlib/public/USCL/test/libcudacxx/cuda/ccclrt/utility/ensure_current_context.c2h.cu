/*
 *
 * Copyright (c) NeXTHub Corporation. All Rights Reserved.
 * DO NOT ALTER OR REMOVE COPYRIGHT NOTICES OR THIS FILE HEADER.
 *
 * Author: Tunjay Akbarli
 * Date: Tuesday, June 11, 2024.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at:
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * Please contact NeXTHub Corporation, 651 N Broad St, Suite 201,
 * Middletown, DE 19709, New Castle County, USA.
 *
 */

#include <uscl/__runtime/ensure_current_context.h>

#include <testing.cuh>

namespace driver = cuda::__driver;

void recursive_check_device_setter(int id)
{
  int cudart_id;
  cuda::__ensure_current_context setter(cuda::device_ref{id});
  CCCLRT_REQUIRE(test::count_driver_stack() == cuda::devices.size() - id);
  auto ctx = driver::__ctxGetCurrent();
  CUDART(hipGetDevice(&cudart_id));
  CCCLRT_REQUIRE(cudart_id == id);

  if (id != 0)
  {
    recursive_check_device_setter(id - 1);

    CCCLRT_REQUIRE(test::count_driver_stack() == cuda::devices.size() - id);
    CCCLRT_REQUIRE(ctx == driver::__ctxGetCurrent());
    CUDART(hipGetDevice(&cudart_id));
    CCCLRT_REQUIRE(cudart_id == id);
  }
}

C2H_TEST("ensure current context", "[device]")
{
  test::empty_driver_stack();
  // If possible use something different than CUDART default 0
  int target_device = static_cast<int>(cuda::devices.size() - 1);

  SECTION("context setter")
  {
    recursive_check_device_setter(target_device);

    CCCLRT_REQUIRE(test::count_driver_stack() == 0);
  }
}
