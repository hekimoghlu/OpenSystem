// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

#include <cub/device/device_reduce.cuh>

#include <uscl/__execution/determinism.h>
#include <uscl/__execution/require.h>
#include <uscl/std/complex>

int main()
{
  namespace stdexec = cuda::std::execution;

  cuda::std::complex<float>* ptr{};
  auto env = cuda::execution::require(cuda::execution::determinism::gpu_to_gpu);

  // expected-error {{"gpu_to_gpu determinism is unsupported"}}
  hipcub::DeviceReduce::Reduce(ptr, ptr, 0, cuda::std::plus<>{}, cuda::std::complex<float>{}, env);
}
